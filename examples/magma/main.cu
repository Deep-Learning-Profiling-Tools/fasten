#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <magma_v2.h>

#define RUNTIME_API_CALL(apiFuncCall)                                        \
  do {                                                                       \
    hipError_t _status = apiFuncCall;                                       \
    if (_status != hipSuccess) {                                            \
      fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
        __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));      \
      exit(-1);                                                              \
    }                                                                        \
  } while (0)

static magma_int_t dev = 0;
static magma_queue_t queue = NULL;
const static int M = 16384;
const static int K = 16;
const static int N = 8;
const static int B = 128;
static float *l_gpu = NULL;
static float *r_gpu = NULL;
static float *p_gpu = NULL;
static float *l_cpu = NULL;
static float *r_cpu = NULL;
static float *p_cpu = NULL;

static magma_int_t m_shapes_cpu[B + 1];
static magma_int_t n_shapes_cpu[B + 1];
static magma_int_t k_shapes_cpu[B + 1];
static magma_int_t *m_shapes_gpu;
static magma_int_t *n_shapes_gpu;
static magma_int_t *k_shapes_gpu;

static magma_int_t l_ldd_cpu[B + 1];
static magma_int_t r_ldd_cpu[B + 1];
static magma_int_t p_ldd_cpu[B + 1];
static magma_int_t *l_ldd_gpu;
static magma_int_t *r_ldd_gpu;
static magma_int_t *p_ldd_gpu;

static float *l_gpu_arr_cpu[B];
static float *r_gpu_arr_cpu[B];
static float *p_gpu_arr_cpu[B];
static float **l_gpu_arr_gpu;
static float **r_gpu_arr_gpu;
static float **p_gpu_arr_gpu;

/*
 *  [16384, 16] x [128, 16, 8] = [16384, 8]

 *  [16384 / 128, 16] x [16, 8]
 *  [16384 / 128, 16] x [16, 8]
 */
static void init_array(float *arr, size_t size) {
  for (size_t i = 0; i < size; ++i) {
    arr[i] = static_cast<float>(i) / size;
  }
}

static void setup() {
  magma_init();
  magma_queue_create(dev, &queue);

  l_cpu = new float[M * K];
  r_cpu = new float[K * N * B];
  p_cpu = new float[M * N];

  RUNTIME_API_CALL(hipMalloc(&l_gpu, sizeof(float) * M * K));
  RUNTIME_API_CALL(hipMalloc(&r_gpu, sizeof(float) * K * N * B));
  RUNTIME_API_CALL(hipMalloc(&p_gpu, sizeof(float) * M * N));

	RUNTIME_API_CALL(hipMalloc(&m_shapes_gpu, sizeof(magma_int_t) * (B + 1)));
	RUNTIME_API_CALL(hipMalloc(&n_shapes_gpu, sizeof(magma_int_t) * (B + 1)));
	RUNTIME_API_CALL(hipMalloc(&k_shapes_gpu, sizeof(magma_int_t) * (B + 1)));

  RUNTIME_API_CALL(hipMalloc(&l_ldd_gpu, sizeof(magma_int_t) * (B + 1)));
  RUNTIME_API_CALL(hipMalloc(&r_ldd_gpu, sizeof(magma_int_t) * (B + 1)));
  RUNTIME_API_CALL(hipMalloc(&p_ldd_gpu, sizeof(magma_int_t) * (B + 1)));

	RUNTIME_API_CALL(hipMalloc(&l_gpu_arr_gpu, sizeof(float *) * B));
	RUNTIME_API_CALL(hipMalloc(&r_gpu_arr_gpu, sizeof(float *) * B));
	RUNTIME_API_CALL(hipMalloc(&p_gpu_arr_gpu, sizeof(float *) * B));
}

static void cleanup() {
  delete [] l_cpu;
  delete [] r_cpu;
  delete [] p_cpu;

  RUNTIME_API_CALL(hipFree(l_gpu));
  RUNTIME_API_CALL(hipFree(r_gpu));
  RUNTIME_API_CALL(hipFree(p_gpu));

	RUNTIME_API_CALL(hipFree(m_shapes_gpu));
	RUNTIME_API_CALL(hipFree(n_shapes_gpu));
	RUNTIME_API_CALL(hipFree(k_shapes_gpu));

	RUNTIME_API_CALL(hipFree(l_ldd_gpu));
	RUNTIME_API_CALL(hipFree(r_ldd_gpu));
	RUNTIME_API_CALL(hipFree(p_ldd_gpu));

	RUNTIME_API_CALL(hipFree(l_gpu_arr_gpu));
	RUNTIME_API_CALL(hipFree(r_gpu_arr_gpu));
	RUNTIME_API_CALL(hipFree(p_gpu_arr_gpu));

  magma_queue_destroy(queue);
  magma_finalize();
}

static void init() {
  init_array(l_cpu, M * K);
  init_array(r_cpu, K * N * B);
  RUNTIME_API_CALL(hipMemcpy(l_gpu, l_cpu, M * K, hipMemcpyHostToDevice));
  RUNTIME_API_CALL(hipMemcpy(r_gpu, r_cpu, K * N * B, hipMemcpyHostToDevice));

  for (size_t i = 0; i < B; ++i) {
    m_shapes_cpu[i] = M / B;
    n_shapes_cpu[i] = N;
    k_shapes_cpu[i] = K;
		// column major
    l_ldd_cpu[i] = m_shapes_cpu[i];
    r_ldd_cpu[i] = K;
    p_ldd_cpu[i] = m_shapes_cpu[i];
    l_gpu_arr_cpu[i] = l_gpu + i * m_shapes_cpu[i] * K;
    r_gpu_arr_cpu[i] = r_gpu + i * K * N;
    p_gpu_arr_cpu[i] = p_gpu + i * m_shapes_cpu[i] * N;
  }

	RUNTIME_API_CALL(hipMemcpy(m_shapes_gpu, m_shapes_cpu, sizeof(magma_int_t) * B, hipMemcpyHostToDevice));
	RUNTIME_API_CALL(hipMemcpy(n_shapes_gpu, n_shapes_cpu, sizeof(magma_int_t) * B, hipMemcpyHostToDevice));
	RUNTIME_API_CALL(hipMemcpy(k_shapes_gpu, k_shapes_cpu, sizeof(magma_int_t) * B, hipMemcpyHostToDevice));

  RUNTIME_API_CALL(hipMemcpy(l_ldd_gpu, l_ldd_cpu, sizeof(magma_int_t) * B, hipMemcpyHostToDevice));
  RUNTIME_API_CALL(hipMemcpy(r_ldd_gpu, r_ldd_cpu, sizeof(magma_int_t) * B, hipMemcpyHostToDevice));
  RUNTIME_API_CALL(hipMemcpy(p_ldd_gpu, p_ldd_cpu, sizeof(magma_int_t) * B, hipMemcpyHostToDevice));

	RUNTIME_API_CALL(hipMemcpy(l_gpu_arr_gpu, l_gpu_arr_cpu, sizeof(float *) * B, hipMemcpyHostToDevice));
	RUNTIME_API_CALL(hipMemcpy(r_gpu_arr_gpu, r_gpu_arr_cpu, sizeof(float *) * B, hipMemcpyHostToDevice));
	RUNTIME_API_CALL(hipMemcpy(p_gpu_arr_gpu, p_gpu_arr_cpu, sizeof(float *) * B, hipMemcpyHostToDevice));
}

static void compute() {
  hipEvent_t start_event, end_event;
  RUNTIME_API_CALL(hipEventCreate(&start_event));
  RUNTIME_API_CALL(hipEventCreate(&end_event));

  RUNTIME_API_CALL(hipEventRecord(start_event));
  magmablas_sgemm_vbatched(
    MagmaNoTrans,
    MagmaNoTrans,
    m_shapes_gpu,
    n_shapes_gpu,
    k_shapes_gpu,
    1.0,
    l_gpu_arr_gpu,
    l_ldd_gpu,
    r_gpu_arr_gpu,
    r_ldd_gpu,
    0.0,
    p_gpu_arr_gpu,
    p_ldd_gpu,
    B,
    queue);
  RUNTIME_API_CALL(hipEventRecord(end_event));
  RUNTIME_API_CALL(hipEventSynchronize(end_event));

  float ms = 0.0;
  RUNTIME_API_CALL(hipEventElapsedTime(&ms, start_event, end_event));
	std::cout << "Elapsed time: " << ms << "ms" << std::endl;
  
  RUNTIME_API_CALL(hipEventDestroy(start_event));
  RUNTIME_API_CALL(hipEventDestroy(end_event));
}

int main() {
  setup();
  init();
  compute();
  cleanup();
  return 0;
}